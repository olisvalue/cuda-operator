#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

// CUDA-ядро: y[i] = x[i] * x[i]
__global__ void square_kernel(
    const float* __restrict__ x,
    float* __restrict__ y,
    size_t n
) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx] * x[idx];
    }
}

void square_cuda(at::Tensor x, at::Tensor y) {
    const size_t n = x.numel();
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    square_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        n
    );
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in square_kernel: %s\n", hipGetErrorString(err));
    }
}
